#include "hip/hip_runtime.h"
#include "wav.h"
#include <fstream>

__global__ static void f_wavConvert(float2* output, short2* input, size_t frames)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	for (int s = offset.x; s < frames; s += stride.x)
	{
		short2 v = input[s];
		output[s] = make_float2(
			v.x / (float)65536,
			v.y / (float)65536);
	}
}
__global__ static void f_wavConvert24(float2* output, uint8_t* input, size_t frames)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	for (int s = offset.x; s < frames; s += stride.x)
	{
		uint32_t a1 = input[s*6+0];
		uint32_t b1 = input[s*6+1];
		uint32_t c1 = input[s*6+2];
		uint32_t v1 = (a1 << 8) | (b1 << 16) | (c1 << 24);

		uint32_t a2 = input[s*6+3];
		uint32_t b2 = input[s*6+4];
		uint32_t c2 = input[s*6+5];
		uint32_t v2 = (a2 << 8) | (b2 << 16) | (c2 << 24);

		int32_t vv1 = (int32_t) v1;
		int32_t vv2 = (int32_t) v2;
		vv1 /= 256;
		vv2 /= 256;
		
		output[s] = make_float2(
			vv1 / (float)(16777216),
			vv2 / (float)(16777216));
	}

}

WavFile::WavFile(const std::string& path) : path(path)
{
	struct hdr_t
	{
		uint32_t chunkId;
		uint32_t chunkSize;
	} header;

	struct fmt_t
	{
		uint16_t audioFormat;
		uint16_t numChannels;
		uint32_t sampleRate;
		uint32_t byteRate;
		uint16_t blockAlign;
		uint16_t bitsPerSample;
	};

	std::ifstream is = std::ifstream(path, std::ifstream::binary);
	is.read((char*)&header, 8);

	char* format = (char*) alloca(4);
	is.read(format, 4);
	assert(!memcmp(format, "WAVE", 4));

	is.read((char*)&header, 8);
	assert(header.chunkSize >= sizeof(fmt_t));
	fmt_t* fmt = (fmt_t*) alloca(header.chunkSize);
	is.read((char*)fmt, header.chunkSize);

#if 0
	Log::info("WAV", "Format: %d", fmt->audioFormat);
	Log::newline("Num Channels: %d", fmt->numChannels);
	Log::newline("Sample Rate: %d", fmt->sampleRate);
	Log::newline("Byte Rate: %d", fmt->byteRate);
	Log::newline("Block Align: %d", fmt->blockAlign);
	Log::newline("Bits per Sample: %d", fmt->bitsPerSample);
#endif

	is.read((char*)&header, 8);
	Log::info("wav", ESC(32;1) "IR [%0.2f s] " ESC(0) ESC(32;2) "%s", 
			header.chunkSize / (float)fmt->byteRate, path.c_str());
	char* hostBuffer = new char[header.chunkSize];
	is.read(hostBuffer, header.chunkSize);

	char* devBuffer;
	int rc = hipMalloc(&devBuffer, header.chunkSize);
	assert(hipSuccess == rc);

	numFrames = header.chunkSize / (fmt->numChannels * (fmt->bitsPerSample >> 3));

	rc = hipMalloc(&buffer, numFrames * sizeof(float2));
	assert(hipSuccess == rc);

	rc = hipMemcpy(devBuffer, hostBuffer, header.chunkSize, hipMemcpyHostToDevice);
	assert(hipSuccess == rc);

	if (6 == fmt->blockAlign && 24 == fmt->bitsPerSample)
	{
		assert(2 == fmt->numChannels);
		f_wavConvert24 <<< 16, 256, 0, 0 >>> ( buffer, (uint8_t*)devBuffer, numFrames);
	}
	else
	{
		assert(2 == fmt->numChannels);
		assert(4 == fmt->blockAlign);
		assert(16 == fmt->bitsPerSample);
		f_wavConvert <<< 16, 256, 0, 0 >>> ( buffer, (short2*)devBuffer, numFrames);
	}
	delete[] hostBuffer;
	
	hipStreamSynchronize(0);
}

#include "hip/hip_runtime.h"
#include "conv.h"

#ifndef CONV_INTERPOLATE
#define CONV_INTERPOLATE 1
#endif

#ifndef CONV_LOWPASS
#define CONV_LOWPASS 1
#endif

__device__ inline hipfftComplex conjugate(hipfftComplex v) { return { v.x, -v.y }; }
__device__ inline hipfftComplex timesj(hipfftComplex v) { return { -v.y, v.x }; }

#if CONV_INTERPOLATE
__global__ static void f_interpolate(
		hipfftComplex* dst, const hipfftComplex* a, const hipfftComplex* b, 
		size_t fftSize, size_t steps, float wet)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	//auto ns = 1.0f / (steps + 5);
	for (int s = offset.x; s < fftSize/2; s += stride.x)
	{
		auto va = a[s];
		auto vb = b[s] * wet;
		auto vd = (vb - va) / (steps + 5); //* ns;
		auto vv = va + vd;
		dst[s] = vv;
		if (s) dst[fftSize - s] = conjugate(vv);
	}
}
#endif

__global__ static void f_unpackC22R(hipfftComplex* L, hipfftComplex* R, const hipfftComplex* src, size_t fftSize)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	//assert(1 == __popcll(fftSize));
	//auto m = fftSize - 1;

	for (auto s = offset.x; s < fftSize/2; s += stride.x)
	{
		auto idxa = s;
		auto idxb = (fftSize - s);

		auto va = src[idxa];
		auto vb = s ? conjugate(src[idxb]) : va;
		auto la = 0.5f * (va + vb);
		auto lb = timesj(-0.5f * (va - vb));

		L[idxa] = la;
		R[idxa] = lb;
		if (s)
		{
			L[idxb] = conjugate(la);
			R[idxb] = conjugate(lb);
		}
	}
}


__global__ static void f_lowpass(hipfftComplex* r, const hipfftComplex* a, size_t fftSize)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	for (auto s = offset.x; s < fftSize/2; s += stride.x)
	{
		auto va = a[s] * cbrt(0.54 - 0.46 * cospif((2.0f * s) / fftSize));
		r[s] = va;
		if (s) r[fftSize - s] = conjugate(va);
	}
}

__global__ static void f_pointwiseAdd(hipfftComplex* r, const hipfftComplex* a, const hipfftComplex* b, size_t n, size_t predelay)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	for (auto s = offset.x; s < n; s += stride.x)
	{
		auto v = a[s];
		if (s >= predelay) v += b[s-predelay];
		r[s] = clamp(v, -1.0f, 1.0f);
	}
}

__global__ static void f_pointwiseMultiplyAndScale(
		hipfftComplex* r, 
		const hipfftComplex* ir1, const hipfftComplex* ir2, 
		const hipfftComplex* a1, const hipfftComplex* a2, 
		size_t n, float scale1, float scale2)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	for (auto s = offset.x; s < n; s += stride.x)
	{
		auto va1 = a1[s];
		auto va2 = a2[s];
		auto vir1 = ir1[s];
		auto vir2 = ir2[s];
		auto re1 = va1.x * vir1.x - va1.y * vir1.y;
		auto re2 = va2.x * vir2.x - va2.y * vir2.y;
		auto im1 = (va1.x + va1.y) * (vir1.x + vir1.y) - re1;
		auto im2 = (va2.x + va2.y) * (vir2.x + vir2.y) - re2;
		r[s] = make_float2(re1, im1) * scale1 + make_float2(re2,im2) * scale2;
	}
}


__global__ static void f_addDryInterleaved(
		hipfftComplex* L, hipfftComplex* R, 
		const hipfftComplex* original, size_t n, 
		float scaleL1, float scaleR1, float scaleL2, float scaleR2)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	for (auto s = offset.x; s < n; s += stride.x)
	{
		auto v = original[s];
		L[s] += v.x * scaleL1 + v.y * scaleL2;
		R[s] += v.x * scaleR1 + v.y * scaleR2;
	}
}

Convolution::Convolution(const std::string& name, uint8_t ccMessage, uint8_t ccStart, size_t fftSize) : 
	JackClient(name),
	_fftSize(fftSize),
	midiIn(nullptr),
	capture{nullptr},
	playback{nullptr}
{
	int rc;
	for (auto i = 0; i<4; i++) 
	{
		rc = hipStreamCreateWithFlags(&_streams[i], hipStreamNonBlocking);
		assert(0 == rc);
	}

	hipfftComplex** pcc[] = {
		&cin, &cin1, &cin2, &cinFFT, 
		&ir.left, &ir.right,
		&irFFT1.left, &irFFT1.right,
		&irFFT2.left, &irFFT2.right,
	};

	for (auto i = 0UL; i < sizeof(pcc) / sizeof(*pcc); i++)
	{
		rc = hipMalloc(pcc[i], fftSize * sizeof(hipfftComplex));
		assert(hipSuccess == rc);
	}
		
	// TODO make this just float
	rc = hipMalloc(&residual.left, (fftSize + CONV_MAX_PREDELAY) * sizeof(hipfftComplex));
	assert(hipSuccess == rc);

	rc = hipMalloc(&residual.right, (fftSize + CONV_MAX_PREDELAY) * sizeof(hipfftComplex));
	assert(hipSuccess == rc);
	
	rc = hipMalloc(&output.left, (fftSize + CONV_MAX_PREDELAY) * sizeof(hipfftComplex));
	assert(hipSuccess == rc);

	rc = hipMalloc(&output.right, (fftSize + CONV_MAX_PREDELAY) * sizeof(hipfftComplex));
	assert(hipSuccess == rc);

	int n[] = { (int)fftSize };
	int inembed[] = { (int)fftSize };
	int istride   = 1;
	int idist     = (int)fftSize;
	int onembed[] = { (int)fftSize };
	int ostride   = 1;
	int odist     = (int)fftSize;
	int batchSize = 1;

	rc = hipfftPlanMany(&_plan, 1, n, 
			inembed, istride, idist,
			onembed, ostride, odist,
			HIPFFT_C2C, batchSize);
	assert(0 == rc);

	cc[0].message  = ccMessage;
	cc[0].select   = ccStart;
	cc[0].predelay = ccStart + 1;
	cc[0].dry      = ccStart + 2;
	cc[0].wet      = ccStart + 3;
	cc[0].speed   = ccStart + 4;
	cc[0].panDry   = ccStart + 5;
	cc[0].panWet  = ccStart + 6;
	cc[0].level  = ccStart + 7;
	
	cc[1].message  = ccMessage + 1;
	cc[1].select   = ccStart;
	cc[1].predelay = ccStart + 1;
	cc[1].dry      = ccStart + 2;
	cc[1].wet      = ccStart + 3;
	cc[1].speed   = ccStart + 4;
	cc[1].panDry   = ccStart + 5;
	cc[1].panWet  = ccStart + 6;
	cc[1].level  = ccStart + 7;
}

void Convolution::onStart()
{
	activate();
	midiIn = addInput("midi_in", JACK_DEFAULT_MIDI_TYPE);
	playback[0] = addOutput("playback_1");
	playback[1] = addOutput("playback_2");
	capture[0] = addInput("capture_1");
	capture[1] = addInput("capture_2");
}

// TODO, make thread safe
void Convolution::prepare(size_t idx, const WavFile& wav, size_t nframes)
{
	int rc;
	
	hipStream_t stream = _streams[0];
	hipfftSetStream(_plan, stream);

	auto buf = _irBuffers[idx];
	if (buf) hipFree(buf);

	// TODO pack real fft into half size buffer
	rc = hipMalloc(&buf, sizeof(hipfftComplex) * (_fftSize << 1));
	assert(hipSuccess == rc);
	
	auto n = min(wav.numFrames, _fftSize - nframes);
	rc = hipMemcpyAsync(buf, wav.buffer, sizeof(hipfftComplex) * n, hipMemcpyDeviceToDevice, stream);
	assert(hipSuccess == rc);
	
	rc = hipfftExecC2C(_plan, buf, buf, HIPFFT_FORWARD);
	assert(hipSuccess == rc);
	
	f_unpackC22R <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, stream >>> (buf, buf+_fftSize, buf,  _fftSize);

	hipStreamSynchronize(stream);
	_irBuffers[idx] = buf;
}

static void handleCC(Convolution::CC& cc, uint8_t m1, uint8_t m2, int v, size_t nb)
{
	if (cc.message == m1)
	{
		if (cc.select == m2) 
		{
			cc.value.select = v * nb / 0x80, cc.value.vsteps = cc.value.speed;
			Log::info("conv", "Selected IR %d", cc.value.select);
		}
		if (cc.predelay == m2) cc.value.predelay = v * CONV_MAX_PREDELAY / 0x80;
		if (cc.dry == m2) cc.value.dry = v / 128.0f;
		if (cc.wet == m2) cc.value.wet = v / 128.0f;
		if (cc.panDry == m2) cc.value.panDry = v / 64.0f - 1;
		if (cc.panWet == m2) cc.value.panWet = v / 64.0f - 1;
		if (cc.level == m2) cc.value.level = v / 128.0f;
		if (cc.speed == m2) 
		{
			cc.value.speed = v * CONV_MAX_SPEED / 0x80;
			if (cc.value.vsteps > cc.value.speed) cc.value.vsteps = cc.value.speed;
		}
	}
}

void Convolution::onProcess(size_t nframes)
{
	int rc;

	auto IN1 = capture[0] ? jack_port_get_buffer(capture[0], nframes) : nullptr;
	auto IN2 = capture[1] ? jack_port_get_buffer(capture[1], nframes) : nullptr;
	auto L = playback[0] ? jack_port_get_buffer(playback[0], nframes) : nullptr;
	auto R = playback[1] ? jack_port_get_buffer(playback[1], nframes) : nullptr;


#if 0
	auto midi = midiIn ? jack_port_get_buffer(midiIn, nframes) : nullptr;
	if (!midi) return;
#endif
	if (!IN1 || !IN2 || !L || !R) return;
#if 0
	auto nevts = jack_midi_get_event_count(midi);
	for (auto i=0UL;i<nevts; i++)
	{
		jack_midi_event_t evt;
		rc = jack_midi_event_get(&evt, midi, i);
		assert(0 == rc);
		
		handleCC(cc[0], evt.buffer[0], evt.buffer[1], evt.buffer[2], _irBuffers.size());
		handleCC(cc[1], evt.buffer[0], evt.buffer[1], evt.buffer[2], _irBuffers.size());

#if 0
		for (auto c=0; c<evt.size; c++) std::cout << std::hex << (int)evt.buffer[c] << " ";
		std::cout << std::endl;
#endif
	}
#endif
	hipEvent_t started, stopped;
	hipEventCreate(&started);
	hipEventCreate(&stopped);
	hipEventRecord(started, _streams[0]);
	hipfftSetStream(_plan, _streams[0]);

	// copy input to device
	rc = hipMemcpy2DAsync(
			cin,  sizeof(hipfftComplex), 
			IN1,  sizeof(float), 
			sizeof(float), nframes,
			hipMemcpyHostToDevice, _streams[1]);
	assert(hipSuccess == rc);

	rc = hipMemcpy2DAsync(
			((float*)cin)+1,  sizeof(hipfftComplex), 
			IN2,  sizeof(float), 
			sizeof(float), nframes,
			hipMemcpyHostToDevice, _streams[1]);
	assert(hipSuccess == rc);

#if CONV_INTERPOLATE
#define CONV_IRFFT1L irFFT1.left
#define CONV_IRFFT1R irFFT1.right
#define CONV_IRFFT2L irFFT2.left
#define CONV_IRFFT2R irFFT2.right

	// Nicer but one of the most time consuming parts
	// interpolate to IR FFT
	f_interpolate <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[2] >>> (
			irFFT1.left, irFFT1.left, _irBuffers[cc[0].value.select], 
			_fftSize, cc[0].value.vsteps, cc[0].value.wet);
	f_interpolate <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[3] >>> (
			irFFT1.right, irFFT1.right, _irBuffers[cc[0].value.select]+_fftSize, 
			_fftSize, cc[0].value.vsteps, cc[0].value.wet);
	if (cc[0].value.vsteps > 0) cc[0].value.vsteps--;

	f_interpolate <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[2] >>> (
			irFFT2.left, irFFT2.left, _irBuffers[cc[1].value.select], 
			_fftSize, cc[1].value.vsteps, cc[1].value.wet);
	f_interpolate <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[3] >>> (
			irFFT2.right, irFFT2.right, _irBuffers[cc[1].value.select]+_fftSize, 
			_fftSize, cc[1].value.vsteps, cc[1].value.wet);
	if (cc[1].value.vsteps > 0) cc[1].value.vsteps--;
#else
#warning Beware. Changing IR during live play may cause some noise.
#define CONV_IRFFT1L (_irBuffers[cc[0].value.select])
#define CONV_IRFFT1R (_irBuffers[cc[0].value.select]+_fftSize)
#define CONV_IRFFT2L (_irBuffers[cc[1].value.select])
#define CONV_IRFFT2R (_irBuffers[cc[1].value.select]+_fftSize)

#endif

	hipStreamSynchronize(_streams[1]);
	
	// get FFT of input
	rc = hipfftExecC2C(_plan, cin, cinFFT, HIPFFT_FORWARD);
	assert(hipSuccess == rc);
	
	f_unpackC22R <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[0] >>> (
			cin1, cin2, cinFFT, _fftSize);

#if CONV_LOWPASS
	f_lowpass <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[0] >>> (
			CONV_IRFFT1L, CONV_IRFFT1L, _fftSize);
	f_lowpass <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[0] >>> (
			CONV_IRFFT1R, CONV_IRFFT1R, _fftSize);
	f_lowpass <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[0] >>> (
			CONV_IRFFT2L, CONV_IRFFT2L, _fftSize);
	f_lowpass <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[0] >>> (
			CONV_IRFFT2R, CONV_IRFFT2R, _fftSize);
#endif
	// multiply ir with input
	float panL1 = cc[0].value.panWet >= 0 ? 1 - cc[0].value.panWet : 1;
	float panR1 = cc[0].value.panWet <= 0 ? 1 + cc[0].value.panWet : 1;
	float panL2 = cc[1].value.panWet >= 0 ? 1 - cc[1].value.panWet : 1;
	float panR2 = cc[1].value.panWet <= 0 ? 1 + cc[1].value.panWet : 1;

	hipStreamSynchronize(_streams[2]);
	f_pointwiseMultiplyAndScale <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[0] >>> (
			output.left, CONV_IRFFT1L, CONV_IRFFT2L, cin1, cin2, _fftSize, 
			1.0f/_fftSize * panL1 * cc[0].value.level, 
			1.0f/_fftSize * panL2 * cc[1].value.level);
	
	hipStreamSynchronize(_streams[3]);
	f_pointwiseMultiplyAndScale <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[0] >>> (
		 	output.right, CONV_IRFFT1R, CONV_IRFFT2R, cin1, cin2, _fftSize, 
			1.0f/_fftSize * panR1 * cc[0].value.level,
			1.0f/_fftSize * panR2 * cc[1].value.level);

	auto tmp = ir;
	// take the inverse FFT of the output
	rc = hipfftExecC2C(_plan, output.left, tmp.left, HIPFFT_BACKWARD);
	assert(hipSuccess == rc);
	rc = hipfftExecC2C(_plan, output.right, tmp.right, HIPFFT_BACKWARD);
	assert(hipSuccess == rc);
		
	// Add the residual
	f_pointwiseAdd <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[0] >>> (
			output.left, residual.left, tmp.left, _fftSize, cc[0].value.predelay);
	
	f_pointwiseAdd <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[0] >>> (
			output.right, residual.right, tmp.right, _fftSize, cc[0].value.predelay);

	// Add dry signal, cin still interleaved
	panL1 = cc[0].value.panDry >= 0 ? 1 - cc[0].value.panDry : 1;
	panR1 = cc[0].value.panDry <= 0 ? 1 + cc[0].value.panDry : 1;
	panL2 = cc[1].value.panDry >= 0 ? 1 - cc[1].value.panDry : 1;
	panR2 = cc[1].value.panDry <= 0 ? 1 + cc[1].value.panDry : 1;
	f_addDryInterleaved <<< 1, CONV_BLOCKSIZE, 0, _streams[0] >>> (
			output.left, output.right, cin, nframes, 
			cc[0].value.dry * panL1 * cc[0].value.level, 
			cc[0].value.dry * panR1 * cc[0].value.level,
			cc[1].value.dry * panL2 * cc[1].value.level, 
			cc[1].value.dry * panR2 * cc[1].value.level);


	// Copy output to host
	rc = hipMemcpy2DAsync(L, sizeof(float), output.left, sizeof(hipfftComplex),
			sizeof(float), nframes, hipMemcpyDeviceToHost, _streams[0]);
	assert(hipSuccess == rc);

	rc = hipMemcpy2DAsync(R, sizeof(float), output.right, sizeof(hipfftComplex),
			sizeof(float), nframes, hipMemcpyDeviceToHost, _streams[0]);
	assert(hipSuccess == rc);
		
	// Copy the residual for next cycle
	rc = hipMemcpyAsync(
			residual.left, 
			output.left + nframes, 
			(_fftSize + CONV_MAX_PREDELAY - nframes) * sizeof(hipfftComplex), 
			hipMemcpyDeviceToDevice, _streams[0]);
	assert(hipSuccess == rc);
	rc = hipMemcpyAsync(
			residual.right, 
			output.right + nframes, 
			(_fftSize + CONV_MAX_PREDELAY - nframes) * sizeof(hipfftComplex), 
			hipMemcpyDeviceToDevice, _streams[0]);
	assert(hipSuccess == rc);

	// Done
	hipEventRecord(stopped, _streams[0]);
	hipEventSynchronize(stopped);

	float elapsed;
	rc = hipEventElapsedTime(&elapsed, started, stopped);
	assert(hipSuccess == rc);

	// initialized nruns to negative value to discard first couple of runs
	if (++_nruns > 0) _runtime += elapsed;
	
	//memcpy(L, in, nframes * sizeof(jack_default_audio_sample_t));
	//memcpy(R, in, nframes * sizeof(jack_default_audio_sample_t));
}


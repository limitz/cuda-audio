#include "hip/hip_runtime.h"
#include "conv.h"

__global__ static void f_interpolate(
		hipfftComplex* dst, hipfftComplex* a, hipfftComplex* b, 
		size_t fftSize, size_t steps, float wet)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	for (int s = offset.x; s < fftSize; s += stride.x)
	{
		auto va = a[s];
		auto vb = b[s] * wet;
		auto vd = (vb - va) / (steps + 5); // just add a little to wet changes
		auto vv = va + vd;
		dst[s] = vv;
	}
}

__device__ inline hipfftComplex conjugate(hipfftComplex v) { return { v.x, -v.y }; }
__device__ inline hipfftComplex timesj(hipfftComplex v) { return { -v.y, v.x }; }

__global__ static void f_unpackC22R(hipfftComplex* L, hipfftComplex* R, hipfftComplex* src, size_t fftSize)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	assert(1 == __popcll(fftSize));
	auto m = fftSize - 1;

	for (auto s = offset.x; s < fftSize/2; s += stride.x)
	{
		auto idxa = s;
		auto idxb = (fftSize - s) & m;

		auto va = src[idxa];
		auto vb = conjugate(src[idxb]);
		auto la = 0.5f * (va + vb);
		auto lb = timesj(-0.5f * (va - vb));

		L[idxa] = la;
		R[idxa] = lb;
		L[idxb] = conjugate(la);
		R[idxb] = conjugate(lb);
	}
}

__global__ static void f_pointwiseAdd(hipfftComplex* r, const hipfftComplex* a, const hipfftComplex* b, size_t n, size_t predelay)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	for (auto s = offset.x; s < n; s += stride.x)
	{
		auto v = a[s];
		if (s >= predelay) v += b[s-predelay];
		r[s] = clamp(v, -1.0f, 1.0f);
	}
}

__global__ static void f_pointwiseMultiplyAndScale(hipfftComplex* r, const hipfftComplex* ir, const hipfftComplex* a, size_t n, float scale)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	for (auto s = offset.x; s < n; s += stride.x)
	{
		auto va = a[s];
		auto vb = ir[s];
		auto re = va.x * vb.x - va.y * vb.y;
		auto im = (va.x + va.y) * (vb.x + vb.y) - re;
		r[s] = make_float2(re, im) * scale;
	}
}

Convolution::Convolution(const std::string& name, uint8_t ccMessage, uint8_t ccStart, size_t fftSize) : 
	JackClient(name),
	_fftSize(fftSize),
	midiIn(nullptr),
	left(nullptr),
	right(nullptr),
	input(nullptr)
{
	int rc;
	for (auto i = 0; i<4; i++) 
	{
		rc = hipStreamCreateWithFlags(&_streams[i], hipStreamNonBlocking);
		assert(0 == rc);
	}


	hipfftComplex** pcc[] = {
		&cin, &cinFFT, 
		&ir.left, &ir.right,
		&irFFT.left, &irFFT.right,
	};

	for (auto i = 0UL; i < sizeof(pcc) / sizeof(*pcc); i++)
	{
		rc = hipMalloc(pcc[i], fftSize * sizeof(hipfftComplex));
		assert(hipSuccess == rc);
	}
		
	// TODO make this just float
	rc = hipMalloc(&residual.left, (fftSize + _maxPredelay) * sizeof(hipfftComplex));
	assert(hipSuccess == rc);

	rc = hipMalloc(&residual.right, (fftSize + _maxPredelay) * sizeof(hipfftComplex));
	assert(hipSuccess == rc);
	
	rc = hipMalloc(&output.left, (fftSize + _maxPredelay) * sizeof(hipfftComplex));
	assert(hipSuccess == rc);

	rc = hipMalloc(&output.right, (fftSize + _maxPredelay) * sizeof(hipfftComplex));
	assert(hipSuccess == rc);

	int n[] = { (int)fftSize };
	int inembed[] = { (int)fftSize };
	int istride   = 1;
	int idist     = (int)fftSize;
	int onembed[] = { (int)fftSize };
	int ostride   = 1;
	int odist     = (int)fftSize;
	int batchSize = 1;

	rc = hipfftPlanMany(&_plan, 1, n, 
			inembed, istride, idist,
			onembed, ostride, odist,
			HIPFFT_C2C, batchSize);
	assert(0 == rc);

	cc.message = ccMessage;
	cc.select = ccStart;
	cc.predelay = ccStart + 1;
	cc.dry = ccStart + 2;
	cc.wet = ccStart + 3;
	cc.isteps = ccStart + 4;
}

void Convolution::onStart()
{
	activate();
	midiIn = addInput("input.midi", JACK_DEFAULT_MIDI_TYPE);
	left = addOutput("output.left");
	right = addOutput("output.right");
	input = addInput("input.mono");
}

// TODO, make thread safe
void Convolution::prepare(size_t idx, const WavFile& wav, size_t nframes)
{
	int rc;
	
	hipStream_t stream = _streams[0];
	hipfftSetStream(_plan, stream);

	auto buf = _irBuffers[idx];
	if (buf) hipFree(buf);

	// TODO pack real fft into half size buffer
	rc = hipMalloc(&buf, sizeof(hipfftComplex) * (_fftSize << 1));
	assert(hipSuccess == rc);
	
	auto n = min(wav.numFrames, _fftSize - nframes);
	rc = hipMemcpyAsync(buf, wav.buffer, sizeof(hipfftComplex) * n, hipMemcpyDeviceToDevice, stream);
	assert(hipSuccess == rc);
	
	rc = hipfftExecC2C(_plan, buf, buf, HIPFFT_FORWARD);
	assert(hipSuccess == rc);
	
	f_unpackC22R <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, stream >>> (buf, buf+_fftSize, buf,  _fftSize);

	hipStreamSynchronize(stream);
	_irBuffers[idx] = buf;
}

void Convolution::onProcess(size_t nframes)
{
	int rc;

	auto in = input ? jack_port_get_buffer(input, nframes) : nullptr;
	auto L = left ? jack_port_get_buffer(left, nframes) : nullptr;
	auto R = right ? jack_port_get_buffer(right, nframes) : nullptr;
	auto midi = midiIn ? jack_port_get_buffer(midiIn, nframes) : nullptr;

	if (!in || !L || !R || !midi) return;

	auto nevts = jack_midi_get_event_count(midi);
	for (auto i=0UL;i<nevts; i++)
	{
		jack_midi_event_t evt;
		rc = jack_midi_event_get(&evt, midi, i);
		assert(0 == rc);
	
#if 0
		for (auto c=0; c<evt.size; c++) std::cout << std::hex << (int)evt.buffer[c] << " ";
		std::cout << std::endl;
#endif

		if (evt.buffer[0] == cc.message)
		{
			if (evt.buffer[1] == cc.select)
			{
				_widx = evt.buffer[2] * _irBuffers.size() / 0x80;
				_interpolationSteps = _maxInterpolationSteps;
				//std::cout << wav[_widx]->path.c_str() << std::endl;
			}
			else if (evt.buffer[1] == cc.predelay)
			{
				_predelay = evt.buffer[2] / 127.0f;
			}
			else if (evt.buffer[1] == cc.dry)
			{
				_dry = evt.buffer[2] / 127.0f;
			}
			else if (evt.buffer[1] == cc.wet)
			{
				_wet = evt.buffer[2] / 127.0f;
			}
			else if (evt.buffer[1] == cc.isteps)
			{
				_maxInterpolationSteps = evt.buffer[2] * 1000 / 0x80 + 1;
				if (_maxInterpolationSteps > _interpolationSteps)
				{
					_interpolationSteps = _maxInterpolationSteps;
				}
			}
		}
	}
	
	hipEvent_t started, stopped;
	hipEventCreate(&started);
	hipEventCreate(&stopped);
	hipEventRecord(started, _streams[0]);
	hipfftSetStream(_plan, _streams[0]);


	// interpolate to IR FFT
	rc = hipMemcpyAsync(ir.left, _irBuffers[_widx], sizeof(hipfftComplex) * _fftSize, 
			hipMemcpyDeviceToDevice, _streams[0]);
	assert(hipSuccess == rc);
	f_interpolate <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[0] >>> (
			irFFT.left, irFFT.left, ir.left, _fftSize, _interpolationSteps, _wet);
	rc = hipMemcpyAsync(ir.right, _irBuffers[_widx]+_fftSize, sizeof(hipfftComplex) * _fftSize, 
			hipMemcpyDeviceToDevice, _streams[0]);
	assert(hipSuccess == rc);
	f_interpolate <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[0] >>> (
			irFFT.right, irFFT.right, ir.right, _fftSize, _interpolationSteps, _wet);
	if (_interpolationSteps > 1) _interpolationSteps--;

	// copy input to device
	rc = hipMemcpy2DAsync(
			cin,  sizeof(hipfftComplex), 
			in,   sizeof(float), 
			sizeof(float), nframes,
			hipMemcpyHostToDevice, _streams[0]);
	assert(hipSuccess == rc);

	// get FFT of input
	rc = hipfftExecC2C(_plan, cin, cinFFT, HIPFFT_FORWARD);
	assert(hipSuccess == rc);

	// multiply ir with input
	f_pointwiseMultiplyAndScale <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[0] >>> (output.left, irFFT.left, cinFFT, _fftSize, 1.0f/_fftSize);
	f_pointwiseMultiplyAndScale <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[0] >>> (output.right, irFFT.right, cinFFT, _fftSize, 1.0f/_fftSize);

	auto tmp = ir;
	// take the inverse FFT of the output
	rc = hipfftExecC2C(_plan, output.left, tmp.left, HIPFFT_BACKWARD);
	assert(hipSuccess == rc);
	rc = hipfftExecC2C(_plan, output.right, tmp.right, HIPFFT_BACKWARD);
	assert(hipSuccess == rc);
		
	// Add the residual
	f_pointwiseAdd <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[0] >>> (output.left, residual.left, tmp.left, _fftSize, (size_t)(_predelay * _maxPredelay));
	f_pointwiseAdd <<< CONV_GRIDSIZE, CONV_BLOCKSIZE, 0, _streams[0] >>> (output.right, residual.right, tmp.right, _fftSize, (size_t)(_predelay * _maxPredelay));
	
	// Copy output to host
	rc = hipMemcpy2DAsync(L, sizeof(float), output.left, sizeof(hipfftComplex),
			sizeof(float), nframes, hipMemcpyDeviceToHost, _streams[0]);
	assert(hipSuccess == rc);
	rc = hipMemcpy2DAsync(R, sizeof(float), output.right, sizeof(hipfftComplex),
			sizeof(float), nframes, hipMemcpyDeviceToHost, _streams[0]);
	assert(hipSuccess == rc);
		
	// Copy the residual for next cycle
	rc = hipMemcpyAsync(
			residual.left, 
			output.left + nframes, 
			(_fftSize + _maxPredelay - nframes) * sizeof(hipfftComplex), 
			hipMemcpyDeviceToDevice, _streams[0]);
	assert(hipSuccess == rc);
	rc = hipMemcpyAsync(
			residual.right, 
			output.right + nframes, 
			(_fftSize + _maxPredelay - nframes) * sizeof(hipfftComplex), 
			hipMemcpyDeviceToDevice, _streams[0]);
	assert(hipSuccess == rc);

	// Done
	hipEventRecord(stopped, _streams[0]);
	hipEventSynchronize(stopped);

	float elapsed;
	rc = hipEventElapsedTime(&elapsed, started, stopped);
	assert(hipSuccess == rc);

	// initialized nruns to negative value to discard first couple of runs
	if (++_nruns > 0) _runtime += elapsed;
	
	//memcpy(L, in, nframes * sizeof(jack_default_audio_sample_t));
	//memcpy(R, in, nframes * sizeof(jack_default_audio_sample_t));
}


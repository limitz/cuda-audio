#include <hipfft/hipfft.h>
#include <ncurses.h>
#include <iostream>
#include <sstream>
#include <fstream>
#include <vector>
#include "gpu.h"
#include "wav.h"
#include "jackclient.h"
#include "conv.h"

#ifndef NUM_CONV_INSTANCES
#define NUM_CONV_INSTANCES 1 
#endif

int main()
{
	selectGpu();


	// Top row of my novation launchcontrol starts at 0x15
	uint8_t ccMessage = 0xB0;
	uint8_t ccStart = 0x15;

	Convolution* instances[NUM_CONV_INSTANCES] = { nullptr };
	for (auto i=0UL; i < NUM_CONV_INSTANCES; i++)
	{
		char* name = (char*)alloca(256);
		sprintf(name, "cudaconv_%lu",i+1);

		// There are 8 controls, let's assume simply that cc is contiguous
		// Other mappings would require changing Convolution::cc member
		auto c = instances[i] = new Convolution(name, ccMessage + i, ccStart);
	
		std::ifstream is("index.txt");
		std::string path;
		for (size_t idx = 0; std::getline(is, path); idx++)
		{
			WavFile w(path);
			c->prepare(idx, w);
		}
		c->start();

		// TODO get connections from settings
		// Connect inputs, assumed to be available
		jack_connect(c->handle, "system:capture_1", jack_port_name(c->capture[0]));
		jack_connect(c->handle, "system:capture_2", jack_port_name(c->capture[1]));
		
		// Connect to stereo output, assumed to be available
		jack_connect(c->handle, jack_port_name(c->playback[0]),  "system:playback_1");
		jack_connect(c->handle, jack_port_name(c->playback[1]), "system:playback_2");

		// Auto connect all MIDI ports
		#if 1
		auto midiports = jack_get_ports(c->handle, NULL, JACK_DEFAULT_MIDI_TYPE, JackPortIsOutput);
		for (auto midiport = midiports; *midiport; midiport++)
		{
			Log::info(__func__, "Found MIDI port: %s", *midiport);
			jack_connect(c->handle, *midiport, jack_port_name(c->midiIn));
		}
		jack_free(midiports);
		#endif

	}

	std::cin.get();

	for (auto i=0UL; i < NUM_CONV_INSTANCES; i++)
	{
		auto c = instances[i];
		if (c->isRunning()) c->stop();
		Log::info(c->name, "Average convolution runtime: %f", c->avgRuntime());
		delete c;
	}

	return 0;
}

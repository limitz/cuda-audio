#include "hip/hip_runtime.h"
#include "audiodevice.h"

#define TAG "AudioDevice"

static int xrunRecovery(snd_pcm_t* pcm, int err)
{
	int rc;
	switch(err)
	{
	case -EPIPE:
		Log::warn(TAG, "XRUN recovery: EPIPE");
		rc = snd_pcm_prepare(pcm);
		if (0 != rc) Log::error(TAG, "Unable to prepare sound device: %d", rc);
		return rc;

	case -ESTRPIPE:
		Log::warn("AudioDevice", "XRUN recovery: ESTRPIPE");
		while ((rc = snd_pcm_resume(pcm)) == -EAGAIN) usleep(100);
		if (rc < 0) 
		{
			rc = snd_pcm_prepare(pcm);
			if (0 != rc) Log::error(TAG, "Unable to prepare sound device: %d", rc);
		}
		return rc;

	default: 
		Log::error("AudioDevice", "XRUN not recoverable: %d", err);
		return err;
	}
}

static int waitForPoll(snd_pcm_t* pcm, struct pollfd *fd, unsigned int count)
{
	unsigned short revents;
	while (true)
	{
		poll(fd, count, -1);
		snd_pcm_poll_descriptors_revents(pcm, fd, count, &revents);
		if (revents & POLLERR)
		{
			Log::error(TAG, "Poll error");
			return -EIO;
		}
		if (revents & POLLOUT)
		{
			return 0;
		}
	}
}

void* AudioDevice::proc(void* context)
{
	int rc;
	AudioDevice* self = reinterpret_cast<AudioDevice*>(context);

	self->_isRunning = true;

	pthread_t thread = pthread_self();
	struct sched_param sched;
	sched.sched_priority = sched_get_priority_max(SCHED_FIFO);
	sched.sched_priority -= (1 + sched.sched_priority / 10);
	pthread_setschedparam(thread, SCHED_FIFO, &sched);
	int r = 0;
	pthread_getschedparam(thread, &r, &sched);
	//doesn't seem to work for now
	//assert(SCHED_FIFO == r);

	bool needsPoll = false;
	while (self->_isRunning)
	{
		size_t requestFrames = self->periodSize*2;

		assert(self->_isOpen);
		if (needsPoll)
		{
			rc = waitForPoll(self->_pcm, self->_poll.fd, self->_poll.count);
			assert(0 == rc);
		}

		auto h = self->handler;
		if (h)
		{
			h->audioDeviceHandlerOnOutputBuffer(self, self->_buffer, requestFrames);
		}

		snd_pcm_uframes_t written = 0;
		while (written < requestFrames)
		{	
			rc = snd_pcm_writei(
					self->_pcm, 
					self->_buffer + written * self->numChannels, 
					min(self->periodSize, requestFrames - written));
			if (rc < 0)
			{
				rc = xrunRecovery(self->_pcm, rc);
				assert(0 == rc);
				break;
			}
			needsPoll = true;
			written += rc;
			if (written >= requestFrames) break;
		
			rc = waitForPoll(self->_pcm, self->_poll.fd, self->_poll.count);
			assert(0 == rc);
		}
	}
	self->_isRunning = false;
	return nullptr;
}

void AudioDevice::start()
{
	int rc;
	assert(!_isOpen);

	rc = snd_pcm_open(&_pcm, deviceId.c_str(), SND_PCM_STREAM_PLAYBACK, 0);
	assert(0 == rc);

	rc = snd_pcm_nonblock(_pcm, 1);
	assert(0 == rc);

	snd_pcm_hw_params_t* hw;
	snd_pcm_hw_params_alloca(&hw);
	snd_pcm_hw_params_any(_pcm, hw);

	rc = snd_pcm_hw_params_set_rate_resample(_pcm, hw, 0);
	assert(0 == rc);

	rc = snd_pcm_hw_params_set_access(_pcm, hw, SND_PCM_ACCESS_RW_INTERLEAVED);
	assert(0 == rc);

	rc = snd_pcm_hw_params_set_format(_pcm, hw, SND_PCM_FORMAT_FLOAT_LE);
	assert(0 == rc);

	rc = snd_pcm_hw_params_set_channels(_pcm, hw, numChannels);
	assert(0 == rc);

	rc = snd_pcm_hw_params_set_rate(_pcm, hw, sampleRate, 0);
	assert(0 == rc);

	//rc = snd_pcm_hw_params_get_buffer_size_min(hw, &bufferSize);
	//assert(0 == rc);

	rc = snd_pcm_hw_params_set_buffer_size(_pcm, hw, bufferSize);
	assert(0 == rc);
	std::cout << bufferSize << std::endl;

	int dir = -1;
	//rc = snd_pcm_hw_params_get_period_size_min(hw, &periodSize, &dir);
	//assert(0 == rc);
	
	rc = snd_pcm_hw_params_set_period_size_near(_pcm, hw, &periodSize, &dir);
	assert(0 == rc);
	std::cout << periodSize << std::endl;

	rc = snd_pcm_hw_params(_pcm, hw);
	assert(0 == rc);

	snd_pcm_sw_params_t* sw;
	snd_pcm_sw_params_alloca(&sw);

	rc = snd_pcm_sw_params_current(_pcm, sw);
	assert(0 == rc);

	rc = snd_pcm_sw_params_set_start_threshold(_pcm, sw, (bufferSize / periodSize) * periodSize);
	assert(0 == rc);

	rc = snd_pcm_sw_params_set_avail_min(_pcm, sw, periodSize);
	assert(0 == rc);

	rc = snd_pcm_sw_params(_pcm, sw);
	assert(0 == rc);

	_poll.count = snd_pcm_poll_descriptors_count(_pcm);
	assert(_poll.count > 0);

	_poll.fd = (struct pollfd*)calloc(_poll.count, sizeof(struct pollfd));
	assert(_poll.fd);

	snd_pcm_poll_descriptors(_pcm, _poll.fd, _poll.count);

	rc = hipHostAlloc(&_buffer, bufferSize * numChannels * sizeof(float), hipHostMallocMapped);
	assert(rc == 0);
	assert(_buffer);

	_isOpen = true;

	rc = pthread_create(&_thread, NULL, AudioDevice::proc, this);
	assert(0 == rc);
}

void AudioDevice::stop()
{
	assert(_isOpen);
	
	if (_isRunning)
	{
		_isRunning = false;
		pthread_join(_thread, 0);
	}

	snd_pcm_drain(_pcm);
	snd_pcm_close(_pcm);
	free(_poll.fd);
	hipFree(_buffer);

	_isOpen = false;
}



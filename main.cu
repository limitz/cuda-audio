#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <iostream>

#include "gpu.h"
#include "audiodevice.h"
#include "mididevice.h"

__global__ static void f_makeTone(hipfftComplex* output, size_t samples, size_t sr, size_t t, float v)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	assert(1 == stride.y);
	assert(0 == offset.y);

	for (auto s = offset.x; s < samples; s += stride.x)
	{
		float a1 = (t + s) % sr;
		float b1 = v * fmaf(powf((sr-a1)/sr,10), fmodf(a1/40,2), -1);
		output[s] = {b1,0};
	}
}

__global__ static void f_makeImpulseResponse(hipfftComplex* output, size_t samples, size_t sr, size_t t, float v)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	assert(1 == stride.y);
	assert(0 == offset.y);

	for (auto s = offset.x; s < samples; s += stride.x)
	{
		output[s] ={s == 1 ? 1.0f : 0, 0};
	}
}

__global__ static void f_pointwiseAdd(hipfftComplex* r, const hipfftComplex* a, const hipfftComplex* b, size_t size)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	for (auto s = offset.x; s < size; s += stride.x)
	{
		auto va = a[s];
		auto vb = b[s];
		r[s] = va;// + vb;
	}
}

__global__ static void f_pointwiseMultiply(hipfftComplex* r, const hipfftComplex* a, const hipfftComplex* b, size_t size)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	for (auto s = offset.x; s < size; s += stride.x)
	{
		auto va = a[s];
		auto vb = b[s];
		auto re = va.x * vb.x - va.y * vb.y;
		auto im = (va.x + va.y) * (vb.x + vb.y) - re;
		r[s] = make_float2(re, im) / size;
	}
}

class MainHandler : public MidiDevice::Handler, public AudioDevice::Handler
{
public:
	MainHandler() :
		_fftSize(0),
		_a(nullptr), _afft(nullptr),
		_b(nullptr), _bfft(nullptr),
		_r(nullptr), _rfft(nullptr)
	{
		int rc;
		for (auto i = 0; i<4; i++) 
		{
			rc = hipStreamCreateWithFlags(&_streams[i], hipStreamNonBlocking);
			assert(0 == rc);
		}
	}

	void prepare(size_t size, size_t channels)
	{
		int rc;
		assert(!_fftSize);
		assert(size > 0);
		assert(channels > 0);

		auto fftSize = size * sizeof(hipfftComplex) * 2;
		rc = hipMalloc(&_a, fftSize);
		assert(0 == rc);
		rc = hipMalloc(&_b, fftSize);
		assert(0 == rc);
		rc = hipMalloc(&_r, fftSize);
		assert(0 == rc);

		rc = hipMalloc(&_afft, fftSize);
		assert(0 == rc);
		rc = hipMalloc(&_bfft, fftSize);
		assert(0 == rc);
		rc = hipMalloc(&_rfft, fftSize);
		assert(0 == rc);
		rc = hipMalloc(&_residual, fftSize);
		assert(0 == rc);

		hipMemset(_residual, 0, fftSize);
		_fftSize = size;
		int n[] = { (int)_fftSize };
		int iembed[] = { (int)_fftSize };
		int oembed[] = { (int)_fftSize };
		int istride = 1;
		int ostride = 1;
		int batch = 1;

		rc = hipfftPlanMany(&_plan, 1, n, 
				iembed, istride, _fftSize, 
				oembed, ostride, _fftSize,
				HIPFFT_C2C, batch);
		assert(0 == rc);
	}	

protected:
	virtual void midiDeviceHandlerOnReceive(MidiDevice* sender, const uint8_t* buffer, size_t len) override
	{
		std::cout << "Received message" << std::endl;
	}

	virtual void audioDeviceHandlerOnOutputBuffer(AudioDevice* sender, float* buffer, size_t frames) override
	{
		int rc;
		static size_t t = 0;
		auto nc = sender->numChannels;
		auto sr = sender->sampleRate;
		hipMemset(_a, 0, _fftSize * sizeof(hipfftComplex));
		f_makeTone <<< 2, 256, 0, _streams[0] >>> (_a, frames, sr, t, 0.15f);
		f_makeImpulseResponse <<< 4, 256, 0, _streams[1] >>> (_b, 4096, sr, 0, 1.0f);
		hipStreamSynchronize(_streams[0]);
		hipStreamSynchronize(_streams[1]);
		hipfftSetStream(_plan, _streams[0]);

		rc = hipfftExecC2C(_plan, _a, _afft, HIPFFT_FORWARD);
		assert(hipSuccess == rc);
		rc = hipfftExecC2C(_plan, _b, _bfft, HIPFFT_FORWARD);
		assert(hipSuccess == rc);
		
		f_pointwiseMultiply <<< 8, 256, 0, _streams[0] >>> (_rfft, _afft, _bfft, _fftSize);
		
		rc = hipfftExecC2C(_plan, _rfft, _r, HIPFFT_BACKWARD);
		assert(hipSuccess == rc);

		f_pointwiseAdd <<< 4, 256, 0, _streams[0] >>> (_a, _r, _residual, _fftSize);

		rc = hipMemcpyAsync(_residual, _r+frames, frames * sizeof(hipfftComplex), hipMemcpyDeviceToDevice, _streams[0]);
		assert(hipSuccess == rc);
		rc = hipMemcpyAsync(buffer, _a, frames*sizeof(hipfftComplex), hipMemcpyDeviceToHost, _streams[0]);
		assert(hipSuccess == rc);

		rc = hipStreamSynchronize(_streams[0]);
		assert(hipSuccess == rc);
		t += frames;
	}

private:
	hipfftHandle _plan;
	hipfftComplex *_a, *_afft;
	hipfftComplex *_b, *_bfft;
	hipfftComplex *_r, *_rfft;
	hipfftComplex *_residual;

	hipStream_t _streams[4];
	size_t _fftSize;

};

int main()
{
	selectGpu();

	MainHandler handler;
	handler.prepare(4096, 2);

	AudioDevice sound("default", &handler);
	sound.start();

	MidiDevice midi("hw:3,0,0");
	midi.start();

	std::cin.get();

	midi.stop();
	sound.stop();
	
	return 0;
}

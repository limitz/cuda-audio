#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <iostream>

#include "gpu.h"
#include "wav.h"
#include "jackclient.h"

static WavFile* wav[8];

__global__ static void f_makeTone(hipfftComplex* output, size_t samples, size_t sr, size_t t, float v)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	for (auto s = offset.x; s < samples; s += stride.x)
	{
		float a1 = (t + s) % sr;
		float b1 = v * fmaf(powf(0.1f+0.9f*(sr-a1)/sr,10), min(a1/200,1.0f) * sinpif(a1/sr * 190) + cospif(a1/sr * 256), 0);
		output[s] = {b1,0};
	}
}

__global__ static void f_deinterleaveIR(hipfftComplex* L, hipfftComplex* R, float2* ir, size_t n)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	for (auto s = offset.x; s < n; s += stride.x)
	{
		auto v = ir[s];
		L[s] = {v.x, 0};
		R[s] = {v.y, 0};
	}
}

__global__ static void f_pointwiseAdd(hipfftComplex* r, const hipfftComplex* a, size_t n)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	for (auto s = offset.x; s < n; s += stride.x)
	{
		auto v = r[s] + clamp(a[s], -1, 1);
		r[s] = v;
	}
}

__global__ static void f_scale(hipfftComplex* r, float scale, size_t n)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	for (auto s = offset.x; s < n; s += stride.x)
	{
		r[s] = clamp(r[s] * scale, -1, 1);
	}
}
__global__ static void f_pointwiseMultiply(hipfftComplex* r, const hipfftComplex* a, size_t n)
{
	auto stride = gridDim * blockDim;
	auto offset = blockDim * blockIdx + threadIdx;

	for (auto s = offset.x; s < n; s += stride.x)
	{
		auto va = a[s];
		auto vb = r[s];
		auto re = va.x * vb.x - va.y * vb.y;
		auto im = (va.x + va.y) * (vb.x + vb.y) - re;
		r[s] = make_float2(re, im);
	}
}

class Convolution : public JackClient
{
public:
	Convolution(size_t fftSize = 96000) : JackClient("Conv"),
		_fftSize(fftSize)
	{
		int rc;
		for (auto i = 0; i<4; i++) 
		{
			rc = hipStreamCreateWithFlags(&_streams[i], hipStreamNonBlocking);
			assert(0 == rc);
		}


		hipfftComplex** cc[] = {
			&cin, &cinFFT, 
			&irFFT.left, &irFFT.right,
			&output.left, &output.right,
			&residual.left, &residual.right
		};
		for (auto i = 0; i < 8; i++)
		{
			rc = hipMalloc(cc[i], fftSize * sizeof(hipfftComplex));
			assert(hipSuccess == rc);
		}

		int n[] = { fftSize };
		int inembed[] = { (int)fftSize };
		int istride   = 1;
		int idist     = (int)fftSize;
		int onembed[] = { (int)fftSize };
		int ostride   = 1;
		int odist     = (int)fftSize;
		int batchSize = 1;

		rc = hipfftPlanMany(&_plan, 1, n, 
				inembed, istride, idist,
				onembed, ostride, odist,
				HIPFFT_C2C, batchSize);
		assert(0 == rc);

		activate();
		midiIn = addInput("input.midi", JACK_DEFAULT_MIDI_TYPE);
		input = addInput("input.mono");
		left = addOutput("output.left");
		right = addOutput("output.right");
	}

	hipfftHandle _plan;
	hipfftComplex* cin;
	hipfftComplex* cinFFT;

	struct
	{
		hipfftComplex* left, *right;
	} irFFT, output, residual;

	JackPort midiIn;
	JackPort input;
	JackPort left;
	JackPort right;

	void onProcess(size_t nframes)
	{
		int rc;
		auto in = jack_port_get_buffer(input, nframes);
		auto L = jack_port_get_buffer(left, nframes);
		auto R = jack_port_get_buffer(right, nframes);


		auto midi = jack_port_get_buffer(midiIn, nframes);
		auto nevts = jack_midi_get_event_count(midi);
		for (auto i=0;i<nevts; i++)
		{
			jack_midi_event_t evt;
			rc = jack_midi_event_get(&evt, midi, i);
			assert(0 == rc);
		
			for (auto c=0; c<evt.size; c++)
			{
				//std::cout << std::hex << (int)evt.buffer[c] << " ";
			}
			//std::cout << std::endl;
			if ((evt.buffer[0] & 0xF0) == 0x90)
			{
				_widx = (_widx + 1) % 8;
			}
		}
		


		hipEvent_t started, stopped;
		hipEventCreate(&started);
		hipEventCreate(&stopped);
		hipEventRecord(started, _streams[0]);
		hipfftSetStream(_plan, _streams[0]);
	
		// move impulse response to irFFT.left , irFFT.right
		f_deinterleaveIR <<< 32, 256, 0, _streams[1] >>> (
				irFFT.left, irFFT.right,
				wav[_widx]->buffer, 
				min(wav[_widx]->numFrames, _fftSize - nframes));

		// copy input to device
		rc = hipMemcpy2DAsync(
				cin,  sizeof(hipfftComplex), 
				in,   sizeof(float), 
				sizeof(float), nframes,
				hipMemcpyHostToDevice, _streams[0]);
		assert(hipSuccess == rc);
		
		// get FFT of input
		rc = hipfftExecC2C(_plan, cin, cinFFT, HIPFFT_FORWARD);
		assert(hipSuccess == rc);

		// await deinterleaveIR
		rc = hipStreamSynchronize(_streams[1]);
		assert(hipSuccess == rc);
		
		// inplace transform irFFT.left and irFFT.right
		rc = hipfftExecC2C(_plan, irFFT.left, output.left, HIPFFT_FORWARD);
		assert(hipSuccess == rc);
		rc = hipfftExecC2C(_plan, irFFT.right, output.right, HIPFFT_FORWARD);
		assert(hipSuccess == rc);

		// multiply ir with input
		f_pointwiseMultiply <<< 64, 256, 0, _streams[0] >>> (output.left, cinFFT, _fftSize);
		f_pointwiseMultiply <<< 64, 256, 0, _streams[0] >>> (output.right, cinFFT, _fftSize);

		// take the inverse FFT of the output
		rc = hipfftExecC2C(_plan, output.left, output.left, HIPFFT_BACKWARD);
		assert(hipSuccess == rc);
		rc = hipfftExecC2C(_plan, output.right, output.right, HIPFFT_BACKWARD);
		assert(hipSuccess == rc);
		f_scale <<< 64, 256, 0, _streams[0] >>> (output.right, _vol * 1.0f/_fftSize, _fftSize);
		f_scale <<< 64, 256, 0, _streams[0] >>> (output.left,  _vol * 1.0f/_fftSize, _fftSize);
		
		// Add the residual
		f_pointwiseAdd <<< 64, 256, 0, _streams[0] >>> (output.left, residual.left, _fftSize - nframes);
		f_pointwiseAdd <<< 64, 256, 0, _streams[0] >>> (output.right, residual.right, _fftSize - nframes);
		
		// Copy output to host
		rc = hipMemcpy2DAsync(L, sizeof(float), output.left, sizeof(hipfftComplex),
				sizeof(float), nframes, hipMemcpyDeviceToHost, _streams[0]);
		assert(hipSuccess == rc);

		rc = hipMemcpy2DAsync(R, sizeof(float), output.right, sizeof(hipfftComplex),
				sizeof(float), nframes, hipMemcpyDeviceToHost, _streams[0]);
		assert(hipSuccess == rc);
		
		// Copy the residual for next cycle
		rc = hipMemcpyAsync(
				residual.left, 
				output.left + nframes, 
				(_fftSize - nframes) * sizeof(hipfftComplex), 
				hipMemcpyDeviceToDevice, _streams[0]);
		assert(hipSuccess == rc);
		rc = hipMemcpyAsync(
				residual.right, 
				output.right + nframes, 
				(_fftSize - nframes) * sizeof(hipfftComplex), 
				hipMemcpyDeviceToDevice, _streams[0]);
		assert(hipSuccess == rc);

		// Done
		hipEventRecord(stopped, _streams[0]);
		rc = hipStreamSynchronize(_streams[0]);
		assert(hipSuccess == rc);

		float elapsed;
		rc = hipEventElapsedTime(&elapsed, started, stopped);
		assert(hipSuccess == rc);

		//memcpy(L, in, nframes * sizeof(jack_default_audio_sample_t));
		//memcpy(R, in, nframes * sizeof(jack_default_audio_sample_t));
	}

	void onShutdown()
	{
	}

private:
	size_t _delay = 1600;
	size_t _lp = 8;
	float _vol = 0.4f;
	size_t _widx = 0;
	size_t _fftSize;
	hipStream_t _streams[4];
};

int main()
{
	selectGpu();

	wav[0] = new WavFile("ir5.wav");
	wav[1] = new WavFile("ir2.wav");
	wav[2] = new WavFile("ir3.wav");
	wav[3] = new WavFile("ir4.wav");
	wav[4] = new WavFile("ir5.wav");
	wav[5] = new WavFile("ir6.wav");
	wav[6] = new WavFile("ir7.wav");
	wav[7] = new WavFile("ir8.wav");

	Convolution c;

	jack_connect(c.handle, "system:capture_1", jack_port_name(c.input));
	jack_connect(c.handle, jack_port_name(c.left),  "system:playback_1");
	jack_connect(c.handle, jack_port_name(c.right), "system:playback_2");
	std::cin.get();

	for (int i=0; i< 8; i++) delete wav[i];
	return 0;
}
